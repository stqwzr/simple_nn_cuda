#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>


__global__ void addVectors(float *vec1, float *vec2, float *result) {
    int index = threadIdx.x;
    result[index] = vec1[index] + vec2[index];
}

int main() {
    const int N = 25 * 32;
    float *vec1_device;
    float *vec2_device;
    float *result_device;

    float *vec1_host = new float[25 * 32];
    float *vec2_host = new float[25 * 32];
    float *result_host = new float[25 * 32];

    hipMalloc(&vec1_device, 25 * 32 * sizeof(float));
    hipMalloc(&vec2_device, 25 * 32 * sizeof(float));
    hipMalloc(&result_device, 25 * 32 * sizeof(float));

    for (int i = 0; i < N; i++) {
        vec1_host[i] = 1.0f;
        vec2_host[i] = 2.0f;
    }

    if (vec1_device == nullptr || vec2_device == nullptr || result_device == nullptr) {
        std::cerr << "Memory allocation failed" << std::endl;
        return 1;
    }

    hipMemcpy(vec1_device, vec1_host, 25 * 32 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(vec2_device, vec2_host, 25 * 32 * sizeof(float), hipMemcpyHostToDevice);

    auto start =     std::chrono::high_resolution_clock::now();

    dim3 num_of_blocks(25);
    dim3 threads_per_block(32);
    addVectors<<<num_of_blocks, threads_per_block>>>(vec1_device, vec2_device, result_device);

    auto end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<float, std::milli> duration_ms = end - start;
    std::cout << "GPU time: " << duration_ms.count() << " ms" << std::endl;

    hipMemcpy(result_device, result_host, 25 * 32 * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(vec1_device);
    hipFree(vec2_device);
    hipFree(result_device);

    return 0;
}
